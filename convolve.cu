#include "hip/hip_runtime.h"
#include <cmath>

#include <boost/gil.hpp>
#include <boost/gil/extension/io/png.hpp>
#include <flash/core.hpp>

#include <iostream>

namespace shino {
std::size_t ceil(std::size_t nominator, std::size_t denominator) {
  return static_cast<std::size_t>(std::ceil(static_cast<double>(nominator) /
                                            static_cast<double>(denominator)));
}

__device__ std::size_t flat_index(std::size_t x, std::size_t y,
                                  std::size_t width) {
  return width * y + x;
}

__global__ void convolve(float *input, float *kernel, float *output,
                         std::size_t width, std::size_t height,
                         std::size_t kernel_width, std::size_t kernel_height) {
  std::size_t origin_x = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t origin_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (origin_y >= height || origin_x >= width) {
    return;
  }

  auto half_k_w = kernel_width / 2;
  auto half_k_h = kernel_height / 2;
  float sum = 0;

  for (std::size_t y = 0; y < kernel_height; ++y) {
    for (std::size_t x = 0; x < kernel_width; ++x) {
      sum += input[flat_index(origin_x + x - half_k_w, origin_y + y - half_k_h,
                              width)] *
             kernel[flat_index(x, y, kernel_width)];
    }
  }

  output[flat_index(origin_x, origin_y, width)] = sum;
}
} // namespace shino

namespace gil = boost::gil;

int main() {
  gil::gray8_image_t molecule_image;
  gil::read_image("gray-molecule.png", molecule_image, gil::png_tag{});
  blaze::DynamicMatrix<float> input(molecule_image.height(),
                                    molecule_image.width());
  flash::to_matrix(gil::view(molecule_image), input);
  std::cout << "is input zero? " << blaze::isZero(input) << '\n';
  float *dev_input = nullptr;
  hipMalloc(&dev_input, sizeof(float) * input.rows() * input.columns());
  hipMemcpy(dev_input, input.data(),
             sizeof(float) * input.rows() * input.columns(),
             hipMemcpyHostToDevice);

  constexpr std::size_t kernel_size = 9;
  float kernel[kernel_size] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
  float *dev_kernel = nullptr;
  hipMalloc(&dev_kernel, sizeof(kernel));
  hipMemcpy(dev_kernel, kernel, sizeof(float) * kernel_size,
             hipMemcpyHostToDevice);

  blaze::DynamicMatrix<float> output(input.rows(), input.columns());
  float *dev_output = nullptr;
  hipMalloc(&dev_output, sizeof(float) * input.rows() * input.columns());

  dim3 threadsPerBlock(32, 32);
  dim3 blocks(30, 40);
  shino::convolve<<<blocks, threadsPerBlock>>>(
      dev_input, dev_kernel, dev_output, input.columns(), input.rows(), 3, 3);

  hipMemcpy(output.data(), dev_output,
             sizeof(float) * output.rows() * output.columns(),
             hipMemcpyDeviceToHost);
  auto mapped = flash::remap_to<gil::uint8_t>(output);
  auto output_image = flash::from_matrix<gil::gray8_image_t>(mapped);
  gil::write_view("convolved-cuda.png", gil::view(output_image),
                  gil::png_tag{});

  std::cout << "is output zero? " << blaze::isZero(output) << '\n';

  /*  gil::gray8_image_t molecule_image;
    gil::read_image("gray-molecule.png", molecule_image, gil::png_tag{});
    gil::gray32f_image_t input_image(molecule_image.dimensions());
    auto input = gil::view(input_image);
    gil::copy_pixels(gil::view(molecule_image), input);

    float *dev_input = nullptr;
    hipMalloc(&dev_input, sizeof(float) * input.size());
    hipMemcpy(dev_input, &(input(0, 0)[0]), sizeof(float) * input.size(),
               hipMemcpyHostToDevice);

    float kernel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
    float *dev_kernel = nullptr;
    hipMalloc(&dev_kernel, sizeof(kernel));
    hipMemcpy(dev_kernel, kernel, sizeof(float) * input.size(),
               hipMemcpyHostToDevice);

    float *dev_output = nullptr;
    hipMalloc(&dev_output, sizeof(float) * input.size());

    dim3 threadsPerBlock(32, 32);
    dim3 blocks(30, 40);
    shino::convolve<<<blocks, threadsPerBlock>>>(
        dev_input, dev_kernel, dev_output, input.width(), input.height(), 3, 3);

    gil::gray32f_image_t output_image(input.dimensions());
    auto output = gil::view(output_image);
    hipMemcpy(&(output(0, 0)[0]), dev_output, sizeof(float) * input.size(),
               hipMemcpyDeviceToHost);
    gil::write_view("convolved.png",
                    gil::color_converted_view<gil::gray8_pixel_t>(output),
                    gil::png_tag{});
  */
}
